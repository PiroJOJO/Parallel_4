#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
//Функция, которая высчитывает разницу между двумя массивами 
__global__ void my_sub(double* arr, double* new_arr, double* c, int n)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;//Индекс для обращения к элементу массива 
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;//Высчитывается из логики - Номер блока на размер блока(кол-во поток) плюс номер потока
    if((i > 0 && i < n-1) && (j > 0 && j < n-1))
    {
        c[i*n + j] = fabs(new_arr[i*n + j] - arr[i*n + j]);//Обращение по индексу по логике преобразования матрицы в одномерный массив
    }
} 
//Функция, которая высчитвает средние значения для обновления сетки
__global__ void update(double* arr, double* new_arr, int n)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    if((i > 0 && i < n-1) && (j > 0 && j < n-1))
    {
        new_arr[i*n + j] = 0.25 * (arr[i*n + j - 1] + arr[i*n + j + 1] + arr[(i - 1)*n + j] + arr[(i + 1)*n + j]);
    }
} 

__global__ void fill(double* arr, double* new_arr, int n)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    arr[0] = new_arr[0] = 10;
    arr[n - 1]= new_arr[n - 1] = 20;
    arr[n * n - 1] = new_arr[n * n - 1] = 30;
    arr[n * (n - 1)] = new_arr[n * (n - 1)] = 20;
    if(i > 0 && i < n-1)
    {
        arr[i] = new_arr[i] = arr[0] + (10.0 / (n-1)) * i;
        arr[n*(n-1) + i] = new_arr[n*(n-1) + i] = arr[n - 1] + 10.0 / (n-1) * i;
        arr[n*i]= new_arr[n*i] = arr[0] + 10.0 / (n-1) * i;
        arr[n*i + n - 1] = new_arr[n*i + n - 1] = arr[n-1] + 10.0 / (n-1) * i;
    }
} 
void swap(double* &a, double* &b)
{
    double *temp = a;
    a = b;
    b = temp;
}
//Функция для просмотра матрицы
void print_matrix(double* vec, size_t n)
{
    for (size_t i = 0; i < n; ++i)
    {
        for (size_t j = 0; j < n; ++j)
        {
            std::cout<<vec[n*i + j]<<' ';
        }
        std::cout<<std::endl;
    }
}

int main(int argc, char *argv[]) {

    auto begin = std::chrono::steady_clock::now();
    if (argc != 7)
    {
        std::cout<<"Enter a string like this: Accuracy _ iterations _ size _"<<std::endl;
    }

    //Считывание значений с командной строки
    double error = std::stod(argv[2]);//Значение ошибки
    size_t iter = std::stoi(argv[4]);//Количество итераций
    size_t n = std::stoi(argv[6]);//Размер сетки 
    //hipSetDevice(0);

    //Объявляем необходимы перменные 
    double* vec = new double[n*n];//Массив для значений на предыдущем шаге
    double* new_vec = new double[n*n];//Массив для значений на текущем шаге
    double* tmp = new double[n*n];//Вспомогаетльный массив для сохранения результата для следующей итерации 
    double* vec_d;
    double* new_vec_d;//Массив для значений на текущем шаге
    double* tmp_d;
    hipMalloc((void **)&vec_d, sizeof(double)*n*n);
    hipMalloc((void **)&new_vec_d, sizeof(double)*n*n);
    hipMalloc((void **)&tmp_d, sizeof(double)*n*n);
    hipMemcpy(vec_d, vec, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(new_vec_d, new_vec, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(tmp_d, tmp, sizeof(double), hipMemcpyHostToDevice);

    double max_error = error + 1; //Объявление максимальной ошибки 
    size_t it = 0;//Счетчик итераций
    // double stepx = 10/(n-1);
    // hipMemcpyToSymbol(HIP_SYMBOL(step), &stepx, sizeof(double));

    //Задаем размер блока и сетки 
    dim3 BLOCK_SIZE = dim3(32, 32);//Размер блока - количество потоков
    dim3 GRID_SIZE = dim3(ceil(n/32.), ceil(n/32.));//Размер сетки - количество блоков
    //Заполнение угловых значений
    //Не забываем, что мы матрицу представляем, как одномерный вектор(вытягиваем ее по по строкам)
    fill<<<GRID_SIZE, BLOCK_SIZE>>>(vec_d, new_vec_d, n);
    hipMemcpy(vec, vec_d, sizeof(double)*n*n, hipMemcpyDeviceToHost);
    hipMemcpy(new_vec, new_vec_d, sizeof(double)*n*n, hipMemcpyDeviceToHost);
 
    //Также инициализируем переменную для расчета максимальной ошибки на cuda
    double* max_errorx;
    hipMalloc(&max_errorx, sizeof(double));

    //Переменные для работы с библиотекой cub
    void* store = NULL;//Доступное устройство выделения временного хранилища. 
    //При NULL требуемый размер выделения записывается в bytes, и никакая работа не выполняется.
    size_t bytes = 0;//Ссылка на размер в байтах распределения store
    hipcub::DeviceReduce::Max(store, bytes, vec, max_errorx, n*n);
    // Allocate temporary storage
	hipMalloc(&store, bytes);
    //Цикл основного алгоритма
    hipStream_t stream;
    hipStreamCreate(&stream);
    hipGraph_t graph;
    hipGraphExec_t instance;

    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    for(int i = 0; i<n*n/4;i+=2)
    {
        update<<<GRID_SIZE,BLOCK_SIZE, 0, stream>>>(new_vec_d, vec_d, n);
        update<<<GRID_SIZE,BLOCK_SIZE, 0, stream>>>(vec_d, new_vec_d, n);
    }
    hipStreamEndCapture(stream, &graph);
    hipGraphInstantiate(&instance, graph, NULL, NULL, 0);

    while(error < max_error && it < iter)
	{        
        it+=n*n/2;
        hipGraphLaunch(instance, stream);
        my_sub<<<GRID_SIZE, BLOCK_SIZE, 0, stream>>>(vec_d, new_vec_d, tmp_d, n);
	    hipcub::DeviceReduce::Max(store, bytes, tmp_d, max_errorx, n*n);
        hipMemcpy(&max_error, max_errorx, sizeof(double), hipMemcpyDeviceToHost);//Обновление ошибки на CPU

    }

    std::cout<<"Error: "<<max_error<<std::endl;
    auto end = std::chrono::steady_clock::now();
    auto elapsed_ms = std::chrono::duration_cast<std::chrono::microseconds>(end-begin);
    // hipDeviceSynchronize();
    // hipMemcpy(vec, vec_d, sizeof(double)*n*n, hipMemcpyDeviceToHost);
    // hipMemcpy(new_vec, new_vec_d, sizeof(double)*n*n, hipMemcpyDeviceToHost);
    // print_matrix(vec, n);
    std::cout<<"time: "<<elapsed_ms.count()<<" mcs\n";
    std::cout<<"Iterations: "<<it<<std::endl;
    delete [] vec; 
    delete [] new_vec;
    hipFree(vec);
    hipFree(new_vec);
    hipFree(tmp);
    return 0;  
}